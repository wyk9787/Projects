#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "util.h"

// The width and height of a sudoku board
#define BOARD_DIM 9

// The width and heigh of a square group in a sudoku board
#define GROUP_DIM 3

// The number of boards to pass to the solver at one time
#define BATCH_SIZE 8192

/**
 * A board is an array of 81 cells. Each cell is encoded as a 16-bit integer.
 * Read about this encoding in the documentation for the next two functions.
 */
typedef struct board { uint16_t cells[BOARD_DIM * BOARD_DIM]; } board_t;

/**
 * Take as input an integer value 0-9 (inclusive) and convert it to the encoded
 * cell form used for solving the sudoku. This encoding uses bits 1-9 to
 * indicate which values may appear in this cell.
 *
 * For example, if bit 3 is set to 1, then the cell may hold a three. Cells that
 * have multiple possible values will have multiple bits set.
 *
 * The input digit 0 is treated specially. This value indicates a blank cell,
 * where any value from one to nine is possible.
 *
 * \param digit   An integer value 0-9 inclusive
 * \returns       The encoded form of digit using bits to indicate which values
 *                may appear in this cell.
 */
__host__ __device__ uint16_t digit_to_cell(int digit) {
  if (digit == 0) {
    // A zero indicates a blank cell. Numbers 1-9 are possible, so set bits 1-9.
    return 0x3FE;
  } else {
    // Otherwise we have a fixed value. Set the corresponding bit in the board.
    return 1 << digit;
  }
}

/*
 * Convert an encoded cell back to its digit form. A cell with two or more
 * possible values will be encoded as a zero. Cells with one possible value
 * will be converted to that value.
 *
 * For example, if the provided cell has only bit three set, this function will
 * return the value 3.
 *
 * \param cell  An encoded cell that uses bits to indicate which values could
 *              appear at this point in the board.
 * \returns     The value that must appear in the cell if there is only one
 *              possibility, or zero otherwise.
 */
__host__ __device__ int cell_to_digit(uint16_t cell) {
// Get the index of the least-significant bit in this cell's value
#if defined(__CUDA_ARCH__)
  int msb = __clz(cell);
  int lsb = sizeof(unsigned int) * 8 - msb - 1;
#else
  int lsb = __builtin_ctz(cell);
#endif

  // Is there only one possible value for this cell? If so, return it.
  // Otherwise return zero.
  if (cell == 1 << lsb)
    return lsb;
  else
    return 0;
}

__host__ __device__ bool update_constraints(uint16_t *target,
                                            uint16_t other_cell) {
  // Decode the other cell value
  int other_digit = cell_to_digit(other_cell);

  // If the other cell is undetermined, no update occurs
  if (other_digit == 0)
    return false;

  // Save the old value of the target cell
  uint16_t old_value = *target;

  // Remove the other cell's value from the target's possibilities
  *target = old_value & ~(1 << other_digit);

  // Did the target cell's possibilities change? If so, return true.
  if (*target != old_value)
    return true;
  else
    return false;
}

__global__ void kernel(board_t *boards) {
  // Threads use their indices as rows and columns
  int col = threadIdx.x;
  int row = threadIdx.y;

  // Put a board in shared memory for the block
  __shared__ board_t board;

  // Each thread copies its cell into the shared board
  int index = col + row * BOARD_DIM;
  board.cells[index] = boards[blockIdx.x].cells[index];

  // Wait for everyone to finish copying
  __syncthreads();

  int changed = 1;

  // Start constraint propagation
  while (__syncthreads_count(changed) > 0) {
    changed = 0;

    // Propagate constraints from other cells in this row
    for (int other_col = 0; other_col < BOARD_DIM; other_col++) {
      // Calculate the index of the other cell in this row
      int other_index = other_col + row * BOARD_DIM;

      // Don't propagate constraints from a cell to itself
      if (index == other_index)
        continue;

      // Update this cell's constraints.
      changed |=
          update_constraints(&board.cells[index], board.cells[other_index]);
    }

    // Propagate constraints from other cells in this column
    for (int other_row = 0; other_row < BOARD_DIM; other_row++) {
      // Calculate the index of the other cell in this col
      int other_index = col + other_row * BOARD_DIM;

      // Don't propagate constraints from a cell to itself
      if (index == other_index)
        continue;

      // Update this cell's constraints
      changed |=
          update_constraints(&board.cells[index], board.cells[other_index]);
    }

    // Propagate constraints for other cells in this group

    // First find the row and column where the current cell's group starts
    int group_row = (row / GROUP_DIM) * GROUP_DIM;
    int group_col = (col / GROUP_DIM) * GROUP_DIM;

    // Loop over cells in the group
    for (int other_row = group_row; other_row < group_row + GROUP_DIM;
         other_row++) {
      for (int other_col = group_col; other_col < group_col + GROUP_DIM;
           other_col++) {
        // Calculate the index of the other cell in this group
        int other_index = other_col + other_row * BOARD_DIM;

        // Don't propagate constraints from a cell to itself
        if (index == other_index)
          continue;

        // Update this cell's constraints
        changed |=
            update_constraints(&board.cells[index], board.cells[other_index]);
      }
    }

    __syncthreads();
    __threadfence_block();
  }

  // Copy shared board data back out to global memory
  boards[blockIdx.x].cells[index] = board.cells[index];
}

void solve_boards(board_t *boards, size_t num_boards) {
  // Allocate space for boards on the GPU
  board_t *gpu_boards;
  if (hipMalloc(&gpu_boards, sizeof(board_t) * num_boards) != hipSuccess) {
    fprintf(stderr, "Failed to allocate boards on GPU.\n");
    exit(2);
  }

  // Copy boards to the GPU
  if (hipMemcpy(gpu_boards, boards, sizeof(board_t) * num_boards,
                 hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy boards to GPU.\n");
    exit(2);
  }

  dim3 threads_per_block(BOARD_DIM, BOARD_DIM);
  dim3 blocks(num_boards);

  kernel<<<blocks, threads_per_block>>>(gpu_boards);

  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Kernel failed.\n");
    exit(2);
  }

  // Copy boards back from the GPU
  if (hipMemcpy(boards, gpu_boards, sizeof(board_t) * num_boards,
                 hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy boards back from GPU.\n");
    exit(2);
  }

  // Free the gpu boards
  if (hipFree(gpu_boards) != hipSuccess) {
    fprintf(stderr, "Failed to free boards on the GPU.\n");
    exit(2);
  }
}

/**
 * Read in a sudoku board from a string. Boards are represented as an array of
 * 81 16-bit integers. Each integer corresponds to a cell in the board. Bits
 * 1-9 of the integer indicate whether the values 1, 2, ..., 8, or 9 could
 * appear in the given cell. A zero in the input indicates a blank cell, where
 * any value could appear.
 *
 * \param output  The location where the board will be written
 * \param str     The input string that encodes the board
 * \returns       true if parsing succeeds, false otherwise
 */
bool read_board(board_t *output, const char *str) {
  for (int index = 0; index < BOARD_DIM * BOARD_DIM; index++) {
    if (str[index] < '0' || str[index] > '9')
      return false;

    // Convert the character value to an equivalent integer
    int value = str[index] - '0';

    // Set the value in the board
    output->cells[index] = digit_to_cell(value);
  }

  return true;
}

/**
 * Print a sudoku board. Any cell with a single possible value is printed. All
 * cells with two or more possible values are printed as blanks.
 *
 * \param board   The sudoku board to print
 */
void print_board(board_t *board) {
  for (int row = 0; row < BOARD_DIM; row++) {
    // Print horizontal dividers
    if (row != 0 && row % GROUP_DIM == 0) {
      for (int col = 0; col < BOARD_DIM * 2 + BOARD_DIM / GROUP_DIM; col++) {
        printf("-");
      }
      printf("\n");
    }

    for (int col = 0; col < BOARD_DIM; col++) {
      // Print vertical dividers
      if (col != 0 && col % GROUP_DIM == 0)
        printf("| ");

      // Compute the index of this cell in the board array
      int index = col + row * BOARD_DIM;

      // Get the index of the least-significant bit in this cell's value
      int digit = cell_to_digit(board->cells[index]);

      // Print the digit if it's not a zero. Otherwise print a blank.
      if (digit != 0)
        printf("%d ", digit);
      else
        printf("  ");
    }
    printf("\n");
  }
  printf("\n");
}

/**
 * Check through a batch of boards to see how many were solved correctly.
 */
void check_solutions(board_t *boards, board_t *solutions, size_t num_boards,
                     size_t *solved_count, size_t *error_count) {

  // Loop over all the boards in this batch
  for (int i = 0; i < num_boards; i++) {
    // Does the board match the solution?
    if (memcmp(&boards[i], &solutions[i], sizeof(board_t)) == 0) {
      // Yes. Record a solved board
      (*solved_count)++;
    } else {
      // No. Make sure the board doesn't have any constraints that rule out
      // values that are supposed to appear in the solution.
      bool valid = true;
      for (int j = 0; j < BOARD_DIM * BOARD_DIM; j++) {
        if ((boards[i].cells[j] & solutions[i].cells[j]) == 0) {
          valid = false;
        }
      }

      // If the board contains an incorrect constraint, record an error
      if (!valid)
        (*error_count)++;
    }
  }
}

int main(int argc, char **argv) {
  // Check arguments
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <input file name>\n", argv[0]);
    exit(1);
  }

  // Try to open the input file
  FILE *input = fopen(argv[1], "r");
  if (input == NULL) {
    fprintf(stderr, "Failed to open input file %s.\n", argv[1]);
    perror(NULL);
    exit(2);
  }

  // Keep track of total boards, boards solved, and incorrect outputs
  size_t board_count = 0;
  size_t solved_count = 0;
  size_t error_count = 0;

  // Keep track of time spent solving
  size_t solving_time = 0;

  // Reserve space for a batch of boards and solutions
  board_t boards[BATCH_SIZE];
  board_t solutions[BATCH_SIZE];

  // Keep track of how many boards we've read in this batch
  size_t batch_count = 0;

  // Read the input file line-by-line
  char *line = NULL;
  size_t line_capacity = 0;
  while (getline(&line, &line_capacity, input) > 0) {
    // Read in the starting board
    if (!read_board(&boards[batch_count], line)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Read in the solution board
    if (!read_board(&solutions[batch_count],
                    line + BOARD_DIM * BOARD_DIM + 1)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Move to the next index in the batch
    batch_count++;

    // Also increment the total count of boards
    board_count++;

    // If we finished a batch, run the solver
    if (batch_count == BATCH_SIZE) {
      size_t start_time = time_ms();
      solve_boards(boards, batch_count);
      solving_time += time_ms() - start_time;

      check_solutions(boards, solutions, batch_count, &solved_count,
                      &error_count);

      // Reset the batch count
      batch_count = 0;
    }
  }

  // Check if there's an incomplete batch to solve
  if (batch_count > 0) {
    size_t start_time = time_ms();
    solve_boards(boards, batch_count);
    solving_time += time_ms() - start_time;

    check_solutions(boards, solutions, batch_count, &solved_count,
                    &error_count);
  }

  // Print stats
  double seconds = (double)solving_time / 1000;
  double solving_rate = (double)solved_count / seconds;

  printf("Boards: %lu\n", board_count);
  printf("Boards Solved: %lu\n", solved_count);
  printf("Errors: %lu\n", error_count);
  printf("Total Solving Time: %lums\n", solving_time);
  printf("Solving Rate: %.2f boards/second\n", solving_rate);

  return 0;
}
